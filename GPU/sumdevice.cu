//------------------------
// Programa en CUDA      -
//------------------------

//----------------
//CPU -> Device
//GPU -> Host
//---------------



//Bibliotecas de entrada y salida 

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

//Predefiniciones 
#define N 512 

__global__  void device_add(int *a, int *b, int *c, int tpb){

        int idx = blockIdx.x*tpb + threadIdx.x;
        int aa = a[idx];
        int bb = b[idx];
    c   [idx] = aa + bb;
}

//------------------------------
// Llenado de Arreglos 
//------------------------------
void fill_array(int *data)
{
    for(int idx = 0; idx < N; idx++)
        data[idx] = idx;
}

//--------------------------------
// Salida de datos 
//--------------------------------
void print_output(int *a, int *b, int *c)
{
    for(int idx = 0; idx < N; idx++)
            printf("\n %d + %d = %d", a[idx], b[idx], c[idx]);
    printf("\n");
}
//---------------------
//  main function 
//---------------------
int main(void) {
    //Arreglos de enteros 
    int *a, *b, *c;
    int *a_device, *b_device, *c_device;
    //Tamaño en memoria de los arreglos 
    int size = N * sizeof(int);
    int threads_per_block = 8;
    int nblocks = N/threads_per_block;

    a = (int *)malloc(size); fill_array(a);
    b = (int *)malloc(size); fill_array(b);
    c = (int *)malloc(size);

    hipMalloc((void **)&a_device, size);
    hipMalloc((void **)&b_device, size);
    hipMalloc((void **)&c_device, size);
    
    hipMemcpy(a_device, a, size, hipMemcpyHostToDevice);
    hipMemcpy(b_device, b, size, hipMemcpyHostToDevice);
    device_add<<<nblocks, threads_per_block >>>(a_device, b_device,c_device, threads_per_block);

    hipMemcpy(c, c_device, size, hipMemcpyDeviceToHost);
    print_output(a,b,c);
    free(a);free(b);free(c);
    hipFree(a_device);hipFree(b_device);hipFree(c_device);
    return 0;

}